
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensor.cu"
#else

// cudaTextureObject_t THCTensor_(getTextureObject)(THCState *state, THCTensor *self)
// {
//   THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
//   cudaTextureObject_t texObj;
//   struct cudaResourceDesc resDesc;
//   memset(&resDesc, 0, sizeof(resDesc));
//   resDesc.resType = cudaResourceTypeLinear;
//   resDesc.res.linear.devPtr = THCTensor_(data)(state, self);
//   resDesc.res.linear.sizeInBytes = THCTensor_(nElement)(state, self) * 4;
//   resDesc.res.linear.desc = cudaCreateChannelDesc(32, 0, 0, 0,
//                                                   cudaChannelFormatKindFloat);
//   struct cudaTextureDesc texDesc;
//   memset(&texDesc, 0, sizeof(texDesc));
//   cudaCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
//   cudaError errcode = cudaGetLastError();
//   if(errcode != cudaSuccess) {
//     if (THCTensor_(nElement)(state, self) > 2>>27)
//       THError("Failed to create texture object, "
//               "nElement:%ld exceeds 27-bit addressing required for tex1Dfetch. Cuda Error: %s",
//               THCTensor_(nElement)(state, self), cudaGetErrorString(errcode));
//     else
//       THError("Failed to create texture object: %s", cudaGetErrorString(errcode));
//   }
//   return texObj;
// }

THC_API int THCTensor_(getDevice)(THCState* state, const THCTensor* tensor) {
  if (!tensor->storage) return -1;
  return THCStorage_(getDevice)(state, tensor->storage);
}

#endif
